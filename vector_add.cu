
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n)
{
	int i = (blockIdx.x*blockDim.x)+threadIdx.x;
	if(i < *d_n){
		printf("i am thread #%d and about to compute c[%d]\n", i, i);
		d_c[i] = d_a[i] + d_b[i];
	}
	else{
		printf("i am thread #%d I am doing nothing\n", i);
	}	
}

int main(){

	int n;

	scanf("%d",&n);

	//declare input and output on host
	int h_a[n];
	int h_b[n];

	for(int i = 0; i < n; i++){
		h_a[i] = i;
		h_b[i] = n-i;
	}

	int h_c[n];

	//part1 copy data from host to device
	int *d_a, *d_b, *d_c, *d_n;
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	//part2 kernel launch
	hipEvent_t start, stop;
	hipEventCreate(&start);	
	hipEventCreate(&stop);

	hipEventRecord(start);
	parallel_vector_add<<<(n+15)/16, 512>>>(d_a, d_b, d_c, d_n);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float millisec = 0;
	hipEventElapsedTime(&millisec, start, stop);
	//part3 copy data from device back to host and free all data allocate on device
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for(int i = 0; i < n; i++)
		printf("%d ", h_c[i]);
	printf("\n");
	printf("Effective time(ms): %f \n",millisec);
}
